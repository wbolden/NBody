#include "hip/hip_runtime.h"
#include "MathUtils.cuh"
#include <cstdio>

#define G 0.0000000000667f //Nm^2/kg^2
#define EP 0.005f
#define EPS EP*EP
#define TIMESTEP 0.01f

__global__ void allPairsNoEP(float3* pos, float3* acc, float* mass, int numPoints, float3* vel)
{
	extern __shared__ float4 bodyInfo[];

	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if(i < numPoints)
	{
		float3 a_i = make_float3(0, 0, 0); //Net acceleration on object i
		float3 posi = pos[i];

		for(int tile = 0; tile < gridDim.x; tile++)
		{
			int localIndex = tile * blockDim.x + threadIdx.x;

			bodyInfo[threadIdx.x] = make_float4(pos[localIndex].x, pos[localIndex].y, pos[localIndex].z, mass[localIndex]);
			__syncthreads();

			for(int j = 0; j < blockDim.x; j++)
			{
				if(i != tile * blockDim.x + j)
				{
					float3 r_ij = bodyInfo[j] - posi;
					float magr = imagnitude(r_ij);

					a_i = a_i + rsqrtf(magr*magr*magr) * bodyInfo[j].w * r_ij;
				}
			}
			__syncthreads();
		}
		acc[i] = a_i * G;

		vel[i] = vel[i] + acc[i] * TIMESTEP;
		pos[i] = pos[i] + vel[i] * TIMESTEP;
	}
}


__global__ void allPairsNormal(float3* pos, float3* acc, float* mass, int numPoints, float3* vel)
{
	extern __shared__ float4 bodyInfo[];

	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if(i < numPoints)
	{
		float3 a_i = make_float3(0, 0, 0); //Net acceleration on object i
		float3 posi = pos[i];

		for(int tile = 0; tile < gridDim.x; tile++)
		{
			int localIndex = tile * blockDim.x + threadIdx.x;

			bodyInfo[threadIdx.x] = make_float4(pos[localIndex].x, pos[localIndex].y, pos[localIndex].z, mass[localIndex]);
			__syncthreads();

			for(int j = 0; j < blockDim.x; j++)
			{
				if(i != tile * blockDim.x + j)
				{	
					float3 r_ij = bodyInfo[j] - posi;

					float dotr = dot(r_ij) + EPS;

					a_i = a_i + rsqrtf(dotr*dotr*dotr) * bodyInfo[j].w * r_ij;
				}
			}
			__syncthreads();
		}
		acc[i] = a_i * G;

		vel[i] = vel[i] + acc[i] * TIMESTEP;
		pos[i] = pos[i] + vel[i] * TIMESTEP;
	}
}


__global__ void integrateEuler(float3* pos, float3* vel, float3* acc, int numPoints)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if(i < numPoints)
	{
		vel[i] = vel[i] + acc[i] * TIMESTEP;
		pos[i] = pos[i] + vel[i] * TIMESTEP;
	}
}

void runPhysics(float3* devPos, float3* devVel, float3* devAcc, float* devMass, int numPoints)
{
	dim3 blockSize = dim3(512);
	dim3 gridSize = dim3((numPoints+blockSize.x-1)/blockSize.x);
	int smem = sizeof(float4)*blockSize.x;

	allPairsNormal<<<gridSize, blockSize, smem>>>(devPos, devAcc, devMass, numPoints, devVel);

//	integrateEuler<<<gridSize, blockSize>>>(devPos, devVel, devAcc, numPoints);

}